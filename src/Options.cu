#include "../include/Options.cuh"

#include <hip/hip_runtime.h>

#include <stdio.h>

extern __constant__ int d_particlesNumber;
extern __constant__ int d_dimensions;
extern __constant__ boxConstraints d_initializationBoxConstraints;
extern __constant__ boxConstraints d_solutionBoxConstraints;
extern __constant__ psoConstants d_psoConstants;
extern __constant__ mcConstants d_mcConstants;

Options::Options(int argc, char* argv[])
{
	if (argc == 3)
	{
		sscanf(argv[1], "%d", &particlesNumber);
		sscanf(argv[2], "%d", &dimensions);
	}
	else
	{
		particlesNumber = 10;
		dimensions = 3;
	}

	if (argc == 4)
		sscanf(argv[3], "%d", &blockSize);
	else
		blockSize = 512;

	initializationBoxConstraints = { -40, 40 };
	solutionBoxConstraints = { -40, 40 };
	float chi = 0.72984f, c1 = 2.05f, c2 = 2.05f;
	psoConstants = { chi, chi * c1, chi * c2 };
	mcConstants = { .1f, .01f };
	task = taskType::TASK_1;
	stopCriterion = 0.01f;
	verbose = true;
	gridSize = (particlesNumber + blockSize - 1) / blockSize;
	logger = false;

	hipMemcpyToSymbol(HIP_SYMBOL(&d_particlesNumber), &particlesNumber, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_dimensions), &dimensions, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_initializationBoxConstraints), &initializationBoxConstraints,
		sizeof(boxConstraints));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_solutionBoxConstraints), &solutionBoxConstraints, sizeof(boxConstraints));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_psoConstants), &psoConstants, sizeof(psoConstants));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_mcConstants), &mcConstants, sizeof(mcConstants));
}