#include "../include/Options.h"

#include <hip/hip_runtime.h>

#include <stdio.h>

extern __constant__ int d_particlesNumber;
extern __constant__ int d_dimensions;
extern __constant__ boxConstraints d_initializationBoxConstraints;
extern __constant__ boxConstraints d_boxConstraints;
extern __constant__ psoConstants d_psoConstants;

Options::Options(int argc, char* argv[])
{
	if (argc == 3)
	{
		sscanf(argv[1], "%d", &particlesNumber);
		sscanf(argv[2], "%d", &dimesions);
	}
	else
	{
		particlesNumber = 10;
		dimesions = 3;
	}

	initializationBoxConstraints = { -40, 40 };
	boxConstraints = { -40, 40 };
	float chi = 0.72984, c1 = 2.05, c2 = 2.05;
	psoConstants = { chi, chi * c1, chi * c2 };

	task = taskType::TASK_1;

	hipMemcpyToSymbol(HIP_SYMBOL(&d_particlesNumber), &particlesNumber, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_dimensions), &dimesions, sizeof(int));
	

	hipMemcpyToSymbol(HIP_SYMBOL(&d_initializationBoxConstraints), &initializationBoxConstraints,
		sizeof(boxConstraints));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_boxConstraints), &boxConstraints, sizeof(boxConstraints));
	hipMemcpyToSymbol(HIP_SYMBOL(&d_psoConstants), &psoConstants, sizeof(psoConstants));

	setBlockSizeInitialization(64);
}

int Options::getBlockSizeInitialization()
{
	return blockSize_initialization;
}

int Options::getGridSizeInitialization()
{
	return gridSize_initialization;
}

void Options::setBlockSizeInitialization( int blockSize )
{
	blockSize_initialization = blockSize;
	gridSize_initialization = (particlesNumber + blockSize - 1) / blockSize;
}