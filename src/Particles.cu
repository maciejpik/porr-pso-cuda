#include "hip/hip_runtime.h"
﻿#include "../include/Particles.cuh"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include<stdio.h>
#include<time.h>

__global__ void _Particles_Initialize_createPrng(int seed, hiprandState* d_prngStates)
{
	int particleId = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, particleId, 0, &d_prngStates[particleId]);
}

Particles::Particles(Options* options) : options(options)
{
	particlesNumber = options->particlesNumber;
	dimensions = options->dimesions;

	hipMalloc(&d_coordinates, particlesNumber * dimensions * sizeof(float));
	hipMalloc(&d_prngStates, particlesNumber * dimensions * sizeof(hiprandState));

	_Particles_Initialize_createPrng << <options->getGridSizeInitialization(), options->getBlockSizeInitialization() >> >
		((int)time(NULL), d_prngStates);
}

Particles::~Particles()
{
	hipFree(d_coordinates);
	hipFree(d_prngStates);
}

void Particles::print()
{
	float* coordinates = new float[particlesNumber * dimensions * sizeof(float)];
	hipMemcpy(coordinates, d_coordinates, particlesNumber * dimensions * sizeof(float),
		hipMemcpyDeviceToHost);

	for (int particleId = 0; particleId < particlesNumber; particleId++)
	{
		printf("[%d] = (", particleId);
		int firstCoord = particleId * dimensions;
		for (int coordinate = 0; coordinate < dimensions; coordinate++)
		{
			if (coordinate != dimensions - 1)
				printf("% .2f,\t", coordinates[firstCoord + coordinate]);
			else
				printf("% .2f)\n", coordinates[firstCoord + coordinate]);
		}
	}
	delete coordinates;
}