﻿#include "include/Options.cuh"
#include "include/PsoParticles.cuh"
#include "include/Pso.cuh"
#include "include/McParticles.cuh"
#include "include/Mc.cuh"

#include <hip/hip_runtime.h>

__constant__ int d_particlesNumber;
__constant__ int d_dimensions;
__constant__ boxConstraints d_initializationBoxConstraints;
__constant__ boxConstraints d_boxConstraints;
__constant__ psoConstants d_psoConstants;
__constant__ mcConstants d_mcConstants;

int main(int argc, char* argv[])
{
	Options* options = new Options(argc, argv);
	options->verbose = false;

	PsoParticles* particles = new PsoParticles(options);
	Pso* pso = new Pso(options, particles);
	pso->solve();

	//McParticles* particles = new McParticles(options);
	//Mc* mc = new Mc(options, particles);
	//mc->solve();
}